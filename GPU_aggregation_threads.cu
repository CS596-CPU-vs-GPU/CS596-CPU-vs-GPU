
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <thread>
#include <vector>
#include <unordered_map>
#include <atomic>
#include <mqueue.h>
#include <fcntl.h>
#include <unistd.h>
#include <chrono> 

#define QUEUE_NAME  "/my_queue"
#define NUM_THREADS 4
#define HASH_SIZE 256

std::atomic<bool> processing_active(true); 

std::vector<std::vector<size_t>> local_buffers(NUM_THREADS);
std::unordered_map<size_t, char*> hashKeyMapping;

uint32_t MurmurHash3_32(const void* key, size_t len, uint32_t seed) {
    const uint8_t* data = (const uint8_t*)key;
    const int nblocks = len / 4;

    uint32_t h1 = seed;
    const uint32_t c1 = 0xcc9e2d51;
    const uint32_t c2 = 0x1b873593;

    const uint32_t* blocks = (const uint32_t*)(data + nblocks * 4);
    for (int i = -nblocks; i; i++) {
        uint32_t k1 = blocks[i];
        k1 *= c1;
        k1 = (k1 << 15) | (k1 >> 17);
        k1 *= c2;

        h1 ^= k1;
        h1 = (h1 << 13) | (h1 >> 19);
        h1 = h1 * 5 + 0xe6546b64;
    }

    const uint8_t* tail = data + nblocks * 4;
    uint32_t k1 = 0;
    switch (len & 3) {
    case 3: k1 ^= tail[2] << 16;
    case 2: k1 ^= tail[1] << 8;
    case 1: k1 ^= tail[0];
        k1 *= c1;
        k1 = (k1 << 15) | (k1 >> 17);
        k1 *= c2;
        h1 ^= k1;
    }

    h1 ^= len;
    h1 ^= h1 >> 16;
    h1 *= 0x85ebca6b;
    h1 ^= h1 >> 13;
    h1 *= 0xc2b2ae35;
    h1 ^= h1 >> 16;

    return h1;
}

void producer_thread_func(mqd_t mq, int thread_id) {
    while (true) {
        char msg[1024];
        ssize_t bytes_read = mq_receive(mq, (char*)&msg, 1024, NULL);
        std::string message(msg, bytes_read);
        if (bytes_read >= 0) {
            size_t hashKey = MurmurHash3_32(msg, strlen(msg), 0);
            hashKeyMapping[hashKey % HASH_SIZE] = strdup(msg); // Note: strdup allocates memory
            local_buffers[thread_id].push_back(hashKey);
        } else {
            std::cerr << "Error receiving message: " << strerror(errno) << std::endl;
            continue;
        }
    }
}

void aggregation_thread_func() {
    while (processing_active) {
        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < NUM_THREADS; ++i) {
            for (size_t hashKey : local_buffers[i]) {
                std::cout << "Aggregated Hash: " << hashKey << std::endl;
            }
            local_buffers[i].clear();
        }

        auto end = std::chrono::high_resolution_clock::now();  
        std::chrono::duration<double> elapsed = end - start;
        std::cout << "Aggregation time: " << elapsed.count() << " seconds." << std::endl;
    }
}

int main() {
    mqd_t mq;
    mq = mq_open(QUEUE_NAME, O_RDONLY);
    if (mq == (mqd_t)-1) {
        perror("mq_open");
        exit(1);
    }

    // Start producer threads
    std::vector<std::thread> producers;
    for (int i = 0; i < NUM_THREADS; ++i) {
        producers.emplace_back(producer_thread_func, mq, i);
    }

    // Start aggregation thread
    std::thread aggregator(aggregation_thread_func);

    // Join threads
    for (auto& producer : producers) {
        producer.join();
    }
    
    aggregator.join();

    return 0;
}

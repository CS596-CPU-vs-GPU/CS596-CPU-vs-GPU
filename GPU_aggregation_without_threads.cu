
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <unordered_map>
#include <vector>
#include <chrono>

#define HASH_SIZE 256

uint32_t MurmurHash3_32(const void* key, size_t len, uint32_t seed) {
    const uint8_t* data = (const uint8_t*)key;
    const int nblocks = len / 4;

    uint32_t h1 = seed;
    const uint32_t c1 = 0xcc9e2d51;
    const uint32_t c2 = 0x1b873593;

    const uint32_t* blocks = (const uint32_t*)(data + nblocks * 4);
    for (int i = -nblocks; i; i++) {
        uint32_t k1 = blocks[i];
        k1 *= c1;
        k1 = (k1 << 15) | (k1 >> 17);
        k1 *= c2;

        h1 ^= k1;
        h1 = (h1 << 13) | (h1 >> 19);
        h1 = h1 * 5 + 0xe6546b64;
    }

    const uint8_t* tail = data + nblocks * 4;
    uint32_t k1 = 0;
    switch (len & 3) {
    case 3: k1 ^= tail[2] << 16;
    case 2: k1 ^= tail[1] << 8;
    case 1: k1 ^= tail[0];
        k1 *= c1;
        k1 = (k1 << 15) | (k1 >> 17);
        k1 *= c2;
        h1 ^= k1;
    }

    h1 ^= len;
    h1 ^= h1 >> 16;
    h1 *= 0x85ebca6b;
    h1 ^= h1 >> 13;
    h1 *= 0xc2b2ae35;
    h1 ^= h1 >> 16;

    return h1;
}

int main() {
    std::ifstream file("input_datasets.json");
    if (!file.is_open()) {
        std::cerr << "Unable to open file: input_datasets.json" << "\n";
        return 1;
    }

    std::string line;
    std::unordered_map<size_t, std::vector<std::string>> hashTable;
    auto start_reading = std::chrono::high_resolution_clock::now(); 

    while (getline(file, line)) {
        uint32_t hashKey = MurmurHash3_32(line.data(), line.length(), 0);
        hashKey %= HASH_SIZE;  // Ensure the hash key is within bounds of HASH_SIZE
        hashTable[hashKey].push_back(line);
    }

    auto end_reading = std::chrono::high_resolution_clock::now();  // End timing reading and inserting
    std::chrono::duration<double> elapsed_reading = end_reading - start_reading;  // Calculate elapsed time
    std::cout << "Time taken for aggregation: " << elapsed_reading.count() << " seconds." << std::endl;

    file.close();

    // Output the results
    for (const auto& [key, values] : hashTable) {
        std::cout << "Hash: " << key << std::endl;
        for (const auto& value : values) {
            std::cout << " - " << value << std::endl;
        }
    }

    return 0;
}

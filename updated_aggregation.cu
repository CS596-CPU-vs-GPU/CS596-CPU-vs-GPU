#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <cstring>
#include <iostream>
#include <fstream>
#include <chrono>
#include "constants.h"
#include <functional>
#include <cstdint>
#include <fcntl.h>
#include <mqueue.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <atomic>
#include <thread>
#include <mutex>
#include <nlohmann/json.hpp>

#define QUEUE_NAME  "/my_queue"

int MAX_SIZE  =  1024;
int TIME_INTERVAL = 300;
int LOCAL_BUFFER_SIZE = 256;

std::mutex fileMutex;
using namespace std::chrono; 
using json = nlohmann::json;

void loadConfigParameters(std::string configString){
   json config =  json::parse(configString);
   TIME_INTERVAL = config["constants"]["TIME_INTERVAL"];
   LOCAL_BUFFER_SIZE = config["constants"]["LOCAL_BUFFER_SIZE"];
   MAX_SIZE = config["constants"]["MAX_SIZE"];
}

uint32_t MurmurHash3_32(const void* key, size_t len, uint32_t seed) {
    const uint8_t* data = (const uint8_t*)key;
    const int nblocks = len / 4;

    uint32_t h1 = seed;
    const uint32_t c1 = 0xcc9e2d51;
    const uint32_t c2 = 0x1b873593;

    // Body
    const uint32_t* blocks = (const uint32_t*)(data + nblocks * 4);
    for (int i = -nblocks; i; i++) {
        uint32_t k1 = blocks[i];

        k1 *= c1;
        k1 = (k1 << 15) | (k1 >> (32 - 15));
        k1 *= c2;

        h1 ^= k1;
        h1 = (h1 << 13) | (h1 >> (32 - 13));
        h1 = h1 * 5 + 0xe6546b64;
    }

    // Tail
    const uint8_t* tail = (const uint8_t*)(data + nblocks * 4);
    uint32_t k1 = 0;

    switch (len & 3) {
        case 3:
            k1 ^= tail[2] << 16;
        case 2:
            k1 ^= tail[1] << 8;
        case 1:
            k1 ^= tail[0];
            k1 *= c1;
            k1 = (k1 << 15) | (k1 >> (32 - 15));
            k1 *= c2;
            h1 ^= k1;
    }

    // Finalization
    h1 ^= len;

    h1 ^= h1 >> 16;
    h1 *= 0x85ebca6b;
    h1 ^= h1 >> 13;
    h1 *= 0xc2b2ae35;
    h1 ^= h1 >> 16;

    return h1;
}

__global__ void countOccurrences(uint32_t* data, int* counts, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        atomicAdd(&counts[data[idx]], 1);
    }
}

void processQueue() {
    mqd_t mq;
    struct mq_attr attr;
    char buffer[MAX_SIZE];
    uint32_t hashBuffer[LOCAL_BUFFER_SIZE];
    int hashCount = 0;

    // Open the message queue
    mq = mq_open(QUEUE_NAME, O_RDONLY);
    if (mq == (mqd_t)-1) {
        std::cerr << "Error opening queue" << std::endl;
        return;
    }

    // Get the attributes of the queue
    mq_getattr(mq, &attr);

    auto start = high_resolution_clock::now();

    // Process messages for a fixed duration
    while (true) {
        ssize_t bytes_read = mq_receive(mq, buffer, MAX_SIZE, nullptr);
        if (bytes_read >= 0) {
            uint32_t hash = MurmurHash3_32(buffer, bytes_read, 42);
            hashBuffer[hashCount++] = hash;

            // If buffer is full, process it
            if (hashCount == LOCAL_BUFFER_SIZE) {
                // Allocate memory on GPU
                uint32_t* d_data;
                int* d_counts;
                int* counts = new int[LOCAL_BUFFER_SIZE]();

                hipMalloc((void**)&d_data, LOCAL_BUFFER_SIZE * sizeof(uint32_t));
                hipMalloc((void**)&d_counts, LOCAL_BUFFER_SIZE * sizeof(int));

                hipMemcpy(d_data, hashBuffer, LOCAL_BUFFER_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
                hipMemcpy(d_counts, counts, LOCAL_BUFFER_SIZE * sizeof(int), hipMemcpyHostToDevice);

                // Launch kernel
                countOccurrences<<<(LOCAL_BUFFER_SIZE + 255) / 256, 256>>>(d_data, d_counts, LOCAL_BUFFER_SIZE);

                hipMemcpy(counts, d_counts, LOCAL_BUFFER_SIZE * sizeof(int), hipMemcpyDeviceToHost);

                // Write counts to file
                {
                    std::lock_guard<std::mutex> lock(fileMutex);
                    std::ofstream outFile("output.txt", std::ios::app);
                    for (int i = 0; i < LOCAL_BUFFER_SIZE; i++) {
                        if (counts[i] > 0) {
                            outFile << i << " " << counts[i] << std::endl;
                        }
                    }
                }

                // Clean up
                hipFree(d_data);
                hipFree(d_counts);
                delete[] counts;

                hashCount = 0; // Reset hash buffer
            }
        }

        auto now = high_resolution_clock::now();
        if (duration_cast<seconds>(now - start).count() >= TIME_INTERVAL) {
            break;
        }
    }

    mq_close(mq);
}

int main() {
    processQueue();
    return 0;
}
